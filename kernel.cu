#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <math.h>

//#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

using std::cout; using std::cin;

#define array_size 20

__device__ float sumtotal;

//void ImpError(hipError_t err);

void ImpError(hipError_t err)
{
	cout << hipGetErrorString(err); // << " en " << __FILE__ << __LINE__;
	//exit(EXIT_FAILURE);
}


__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
		C[i] = A[i] + B[i];
}


void vecAdd(float* A, float* B, float* C, int n)
{
	int size = n * sizeof(float);
	float* d_A, * d_B, * d_C;

	hipError_t err = hipSuccess;

	err = hipMalloc((void**)& d_A, size);

	if (err != hipSuccess)
	{
		cout << "d_A";
		ImpError(err);
	}


	err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
		ImpError(err);

	err = hipMalloc((void**)& d_B, size);

	if (err != hipSuccess)
		ImpError(err);

	err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
		ImpError(err);

	err = hipMalloc((void**)& d_C, size);

	if (err != hipSuccess)
		ImpError(err);

	//<<#bloques,#threads por bloques>>
	vecAddKernel << <ceil(n / 512.0), 512 >> > (d_A, d_B, d_C, n);

	err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
		ImpError(err);

	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


// Implementacion simple fig 5.13
__global__ void SimpleReduce(float* vec_x, float* sum)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float partialSum[array_size];
	partialSum[threadIdx.x]	= vec_x[i];

	unsigned int t = threadIdx.x;
	

	//if (t < array_size)
	//{
		for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
		{

			__syncthreads();
			if ( (t % (2 * stride) == 0) && (t+stride) < array_size  )
			{
				partialSum[t] += partialSum[t + stride];
			}
		}	
	//}

		__syncthreads();


	// write result for this block to global mem
	//if (tid == 0)
		//g_odata[blockIdx.x] = sdata[0];
	if (threadIdx.x == 0)
	{
		sum[0] = partialSum[0];
		sumtotal = partialSum[0];
	}
	
}


__global__ void Reduce(unsigned int* g_odata, unsigned int* g_idata, unsigned int len) {
	extern __shared__ unsigned int sdata[];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	sdata[tid] = 0;

	if (i < len)
	{
		sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
	}

	__syncthreads();

	// do reduction in shared mem
	// this loop now starts with s = 512 / 2 = 256
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce4(unsigned int* g_odata, unsigned int* g_idata, unsigned int len) {
	extern __shared__ unsigned int* sdata;

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	sdata[tid] = 0;

	if (i < len)
	{
		sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
	}

	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid < 32)
	{
		sdata[tid] += sdata[tid + 32];
		sdata[tid] += sdata[tid + 16];
		sdata[tid] += sdata[tid + 8];
		sdata[tid] += sdata[tid + 4];
		sdata[tid] += sdata[tid + 2];
		sdata[tid] += sdata[tid + 1];
	}

	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

void vecSum(float* A, float* sum)
{
	int size = array_size * sizeof(float);
	float* d_A, *d_sum;


	hipError_t err = hipSuccess;

	err = hipMalloc((void**)& d_A, size);
	err = hipMalloc((void**)& d_sum, sizeof(float));

	if (err != hipSuccess)
	{
		cout << "d_A";
		ImpError(err);
	}

	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	//<<#bloques,#threads por bloques>>
	SimpleReduce << <ceil(array_size / 512.0), 512 >> > (d_A, d_sum);

	//err = hipMemcpy(sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		cout << "aqui";
		ImpError(err);
	}
		

	hipFree(d_A);
}


void Imprimir(float* A, int n)
{
	for (int i = 0; i < n; ++i)
		if (i < n) cout << A[i] << " ";
	cout << "\n";
}

void GenVector(float* A, int n)
{

	for (int i = 0; i < n; ++i)
		A[i] = static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / n));
}


int main(int argc, char** argv)
{
	//int array_size = 10;


	float* A, *sum;
	srand(time(NULL));
	/*
	if (argc == 2)
	{
		array_size = strtof(argv[1], NULL);
	}
	else
		cout << "Ingrese array_size"; cin >> array_size;
	*/

	A = new float[array_size];
	sum = new float[1]{ 0 };

	GenVector(A, array_size);

	vecSum(A, sum);

	Imprimir(A, array_size);

	cout << "suma: " << sum[0];
	cout << "suma total:" << sumtotal;
	

	return 0;
}